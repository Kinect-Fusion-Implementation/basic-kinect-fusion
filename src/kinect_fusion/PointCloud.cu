#include "hip/hip_runtime.h"
#include "PointCloud.h"
#include <iostream>

__global__ void computeVerticesKernel(float *depthMap, Vector3f *vertexMap, const Matrix3f depthIntrinsics, const Matrix4f depthExtrinsics, const unsigned int width, const unsigned int height, int level, float minf)
{
	unsigned w = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned h = blockIdx.y * blockDim.y + threadIdx.y;

	if (w > width || h > height)
	{
		printf("returning for: w= %i , h = %i", w, h);
		return;
	}

	// Get depth intrinsics.
	float fovX = depthIntrinsics(0, 0) / pow(2, level);
	float fovY = depthIntrinsics(1, 1) / pow(2, level);
	float cX = depthIntrinsics(0, 2) / pow(2, level);
	float cY = depthIntrinsics(1, 2) / pow(2, level);

	// Compute inverse depth extrinsics.
	Matrix4f depthExtrinsicsInv = depthExtrinsics.inverse();
	Matrix3f rotationInv = depthExtrinsicsInv.block(0, 0, 3, 3);
	Vector3f translationInv = depthExtrinsicsInv.block(0, 3, 3, 1);

	// For every pixel row.
	unsigned int idx = h * width + w; // linearized index
	float depth = depthMap[idx];
	if (depth == minf)
	{
		vertexMap[idx] = Vector3f(minf, minf, minf);
	}
	else
	{
		// Back-projection to camera space.
		vertexMap[idx] = rotationInv * Vector3f((w - cX) / fovX * depth, (h - cY) / fovY * depth, depth) + translationInv;
	}
}

__global__ void computeNormalsKernel(float *depthMap, Vector3f *vertexMap, Vector3f *normalMap, const unsigned int width, const unsigned int height, float minf, const unsigned int maxDistance = 10)
{
	unsigned w = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned h = blockIdx.y * blockDim.y + threadIdx.y;

	if (w > width || h > height)
	{
		printf("returning for: w= %i , h = %i", w, h);
		return;
	}

	const float maxDistanceHalved = maxDistance / 2.f;
	unsigned int idx = h * width + w; // linearized index

	// We need to compute derivatives and then the normalized normal vector (for valid pixels).
	if (h == 0 || w == 0 || w == (height - 1) || h == (height - 1))
	{
		normalMap[idx] = Vector3f(minf, minf, minf);
		return;
	}

	const float du = 0.5f * (depthMap[idx + 1] - depthMap[idx - 1]);
	const float dv = 0.5f * (depthMap[idx + width] - depthMap[idx - width]);

	if (!isfinite(du) || !isfinite(dv) || abs(du) > maxDistanceHalved || abs(dv) > maxDistanceHalved)
	{
		normalMap[idx] = Vector3f(minf, minf, minf);
	}
	else
	{
		// TODO: Compute the normals using central differences.
		normalMap[idx] = (vertexMap[idx + width] - vertexMap[idx - width]).cross(vertexMap[idx + 1] - vertexMap[idx - 1]);
		normalMap[idx].normalize();
	}
}

__host__ PointCloud::PointCloud(float *depthMap, const Matrix3f &depthIntrinsics, const Matrix4f &depthExtrinsics, const unsigned int width, const unsigned int height, int level, const unsigned int maxDistance) : m_width(width), m_height(height)
{
	// The provided depthmap should already be located on the DEVICE
	dim3 threadBlocks(20, 20);
	dim3 blocks(width / 20, height / 20);
	size_t m_memorySize = sizeof(Vector3f) * width * height;
	m_depthMap = depthMap;
	
	//float *depth;
	//hipMalloc(&depth, width * height * sizeof(float));
	//hipMemcpy(depth, depthMap, width * height * sizeof(float), hipMemcpyHostToDevice);
	//m_depthMap = depth;

	hipMalloc(&m_points, m_memorySize);
	hipMalloc(&m_normals, m_memorySize);

	computeVerticesKernel<<<blocks, threadBlocks>>>(m_depthMap, m_points, depthIntrinsics, depthExtrinsics, width, height, level, MINF);

	// Set flag that the device memory still has to be copied
	m_pointsOnCPU = false;
	m_normalsOnCPU = false;
}

__host__ PointCloud::~PointCloud()
{
	hipFree(m_points);
	hipFree(m_normals);
	hipFree(m_depthMap);
	free(m_points_cpu);
	free(m_normals_cpu);
}

__host__ Vector3f *PointCloud::getPointsCPU()
{
	std::cout << "Copy pointcloud to cpu." << std::endl;
	if (!m_pointsOnCPU)
	{
		// Allocate memory for copy
		m_points_cpu = new Vector3f[m_width * m_height];
		hipMemcpy(m_points_cpu, m_points, m_width * m_height * sizeof(Vector3f), hipMemcpyDeviceToHost);
		m_pointsOnCPU = true;
	}
	return m_points_cpu;
}

__host__ Vector3f *PointCloud::getNormalsCPU()
{
	if (!m_normalsOnCPU)
	{
		// Allocate memory for copy
		m_normals_cpu = new Vector3f[m_width * m_height];
		hipMemcpy(m_normals_cpu, m_normals, m_width * m_height * sizeof(Vector3f), hipMemcpyDeviceToHost);
		m_normalsOnCPU = true;
	}

	return m_normals_cpu;
}